#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <stdbool.h>

#define eps 1e-2
#define TILE_WIDTH 32  //89*89*4*2=63368B<64KB

int m,n,k;
double duration;

__global__ void MatrixMulKernle(int m, int n, int k, float *A,float  *B, float *C)
{
	__shared__ float ds_A[TILE_WIDTH][TILE_WIDTH]; 
	__shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
 
	int bx = blockIdx.x;		int by = blockIdx.y;
	int tx = threadIdx.x;		int ty = threadIdx.y;
 
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
 
	float Cvalue = 0;
 
	for (int t=0; t<(n-1)/TILE_WIDTH+1; ++t)
	{
		if (Row < m && t * TILE_WIDTH + tx < n)
		    ds_A[tx][ty] = A[Row*n+t*TILE_WIDTH+tx];
		else
			ds_A[tx][ty] = 0.0;
 
		if (t * TILE_WIDTH + ty < n && Col < k)
            ds_B[tx][ty] = B[(t*TILE_WIDTH + ty)*k+Col];
		else
			ds_B[tx][ty] = 0.0;	
 
		__syncthreads();
		
		for (int i = 0; i < TILE_WIDTH; ++i)
            Cvalue += ds_A[i][ty] * ds_B[tx][i];
 
		__syncthreads();
 
		if(Row < m && Col < k)
			C[Row*k+Col]=Cvalue;		
	}
}

float ds_A[TILE_WIDTH][TILE_WIDTH]; 
float ds_B[TILE_WIDTH][TILE_WIDTH];
void MatrixMulKernleSerialize(int m, int n, int k, float *A,float  *B, float *C)
{
	for(int bx=0;bx<(k-1)/TILE_WIDTH+1;bx++)
	{
		for(int by=0;by<(m-1)/TILE_WIDTH+1;by++)
		{
			for (int t=0; t<(n-1)/TILE_WIDTH+1; ++t)
			{
				for(int tx=0;tx<TILE_WIDTH;tx++)
				for(int ty=0;ty<TILE_WIDTH;ty++)
				{
					int Row = by * TILE_WIDTH + ty;
					int Col = bx * TILE_WIDTH + tx;

					if (Row < m && t * TILE_WIDTH + tx < n)
						ds_A[tx][ty] = A[Row*n+t*TILE_WIDTH+tx];
					else
						ds_A[tx][ty] = 0.0;
			
					if (t * TILE_WIDTH + ty < n && Col < k)
						ds_B[tx][ty] = B[(t*TILE_WIDTH + ty)*k+Col];
					else
						ds_B[tx][ty] = 0.0;	
				}
				for(int tx=0;tx<TILE_WIDTH;tx++)
				for(int ty=0;ty<TILE_WIDTH;ty++)
				{
					int Row = by * TILE_WIDTH + ty;
					int Col = bx * TILE_WIDTH + tx;

					float Cvalue = 0;
	
					for (int i = 0; i < TILE_WIDTH; ++i)
						Cvalue += ds_A[i][ty] * ds_B[tx][i];
			
					if(Row < m && Col < k)
						C[Row*k+Col]+=Cvalue;		
				}
			}
		}
	}
}

void MatrixMulSample(int m, int n, int k, float* A, float* B, float* C)
{
    for (int Row = 0; Row < m; ++ Row)
		for (int Col = 0; Col < k; ++ Col )
		{
			float sum = 0;
			for (int i = 0; i < n; ++i)
			{
				float a = A[Row * n + i];
				float b = B[Col + i * k];
				sum += a * b;
			}
			C[Row* k + Col] = sum;
		}
}
 
int main()
{
	clock_t start, stop;
	srand(time(0));
	m=1001,n=100003,k=987;

	float *A=(float*)malloc(m*n*sizeof(float));
	float *B=(float*)malloc(n*k*sizeof(float));
	float *C=(float*)malloc(m*k*sizeof(float));
	float *sample_C=(float*)malloc(m*k*sizeof(float));

	for(int i=0;i<m*n;i++)
		A[i]=(float)(rand()%100)/100;
	for(int i=0;i<n*k;i++)
		B[i]=(float)(rand()%100)/100;

	//分配显存空间
	int size = sizeof(float);
	float *d_a;
	float *d_b;
	float *d_c;
	hipMalloc((void**)&d_a,m*n*size);
	hipMalloc((void**)&d_b,n*k*size);
	hipMalloc((void**)&d_c,m*k*size);
 
	//把数据从Host传到Device
	hipMemcpy(d_a, A, size*m*n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B, size*n*k, hipMemcpyHostToDevice);
	hipMemcpy(d_c, C, size*m*k, hipMemcpyHostToDevice);
 
	//分配网格结构
	dim3 dimGrid((k-1)/TILE_WIDTH+1,(m-1)/TILE_WIDTH+1,1);	//向上取整
	dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);
 
	start=clock();
	//调用内核函数
	MatrixMulKernle<<<dimGrid,dimBlock>>>(m,n,k,d_a,d_b,d_c);
	hipDeviceSynchronize();
	//将结果传回到主机端
	stop=clock();
	duration=(double)(stop-start)/CLK_TCK;
	printf("GPU version time used    : %.3lfs\n",duration);

	hipMemcpy(C, d_c, size*m*k, hipMemcpyDeviceToHost);

	// start=clock();  //C:k*m 
	// MatrixMulKernleSerialize(m,n,k,A,B,C);
	// stop=clock();
	// duration=(double)(stop-start)/CLK_TCK;
	// printf("Tile version time used:   %.3lfs\n",duration);

	start=clock();
	MatrixMulSample(m,n,k,A,B,sample_C);
	stop=clock();
	duration=(double)(stop-start)/CLK_TCK;
	printf("Normal version time used : %.3lfs\n",duration);

	bool flag=1;
	for (int i=0;i<m*k;i++)
		if(fabs(C[i]-sample_C[i])>eps)
		{
			flag=0;
			printf("%f %f\n",C[i],sample_C[i]);
		}
	if(flag)
		puts("PASS");
	else
		puts("FAILED");

	free(A);
	free(B);
	free(C);
	free(sample_C);

    //释放显存空间
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
 
	return 0;
}